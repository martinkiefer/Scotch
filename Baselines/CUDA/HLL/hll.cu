#include "hip/hip_runtime.h"
#pragma GCC diagnostic ignored "-Wignored-attributes"
#include <iostream>
#include <chrono>

#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_int_distribution.hpp>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ unsigned int parity(unsigned int x) {
    unsigned int y;
    y = x ^ (x >> 1);
    y = y ^ (y >> 2);
    y = y ^ (y >> 4);
    y = y ^ (y >> 8);
    y = y ^ (y >>16);
    return y & 1;
 }
 
 __device__ unsigned int is_set(unsigned int x, unsigned int pos) {
     return (x >> pos) & 1;
 }
 

 __global__ void construct_sketch(
    unsigned int replicas,
    unsigned int skn_cols,
    unsigned int n_values,
    unsigned int* __restrict__ c0,
    unsigned int* __restrict__ c0_select_seed,
    unsigned int* __restrict__ c0_update_seed,
    unsigned int* __restrict__ sketches) 
{
    unsigned int global_size = gridDim.x * blockDim.x;
    unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;


    //unsigned int my_c0_ls = (sketch < replicas) ? c0_ls[sketch] : 0; 
    //unsigned int my_c0_ss = (sketch < replicas) ? is_set(c0_ss[sketch/32], sketch % 32) : 0;
    //unsigned int* my_c0_select_seed = (sketch < replicas) ? c0_select_seed + sketch : 0;

    unsigned int r = global_id % replicas;

    unsigned int sseeds[32];
    for(int i = 0; i < 32; i++){
        sseeds[i] = c0_select_seed[i];
    }

    unsigned int useeds[32];
    for(int i = 0; i < 32; i++){
        useeds[i] = c0_update_seed[i];
    }

    //Energy consumption endless loop. Comment in, if necessary.
    //while(1)
    for(unsigned int i = global_id; i < n_values; i += global_size){
            unsigned int select = 0;
            for(int k = 0; k < 32; k++)  if(is_set(c0[i],k)) select ^= sseeds[k] ;
            select = select % skn_cols;

            unsigned int update = 0;
            for(int k = 0; k < 32; k++)  if(is_set(c0[i],k)) update ^= useeds[k] ;
            //printf("%d\n", __clz(update));
            //int update = ech3(c0[i],my_c0_ls,my_c0_ss);
            atomicMax(&sketches[r*skn_cols+select], (unsigned int) (__clz(update)+1));
            //printf(" %d %d\n",__clz(update)+1, sketches[r*skn_cols+select]);
    }
}

typedef struct{

    size_t replicas;
    size_t skn_cols;

    unsigned int* sk_t0;
    unsigned int ts0;

    unsigned int* c0;
    unsigned int* c0_select_seed;
    unsigned int* c0_update_seed;

} parameters;

void* cudaAllocAndCopy(void* hst_ptr, size_t size){
    void* d_ptr;
    hipMalloc((void **) &d_ptr, size);
    hipMemcpy(d_ptr, hst_ptr, size, hipMemcpyHostToDevice);
    return d_ptr;
}

void writeSArrayToFile(const char* filename, int* elements, size_t size){
    FILE *f1 = fopen(filename, "w");
    assert(f1 != NULL);
    
    fwrite(elements, sizeof(int), size, f1);
    fclose(f1);
}

unsigned int* readUArrayFromFile(const char* filename, size_t * filesize = NULL){
    FILE *f1 = fopen(filename, "rb");
    assert(f1 != NULL);
    fseek(f1, 0, SEEK_END);
    size_t fsize1 = ftell(f1);
    if(filesize) *filesize=fsize1;
    fseek(f1, 0, SEEK_SET);
    unsigned int* tab1 = (unsigned int*) malloc(fsize1);
    size_t x = fread(tab1, fsize1, 1, f1);
    fclose(f1);

    return tab1;
}

double sketch_contruction(parameters* p){
    size_t local = 64;
    int tot_SM = 0;
    int tot_tpsm = 0;
    hipDeviceGetAttribute(&tot_SM, hipDeviceAttributeMultiprocessorCount, 0);
    hipDeviceGetAttribute(&tot_tpsm, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
    unsigned int target_utilization = tot_SM*tot_tpsm;
    size_t global = target_utilization;

    auto begin = std::chrono::high_resolution_clock::now();
    int iterations = 1;
    for(int i = 0; i < iterations; i++){
            construct_sketch<<<global/local, local>>>((unsigned int) p->replicas, (unsigned int) p->skn_cols, p->ts0, p->c0, p->c0_select_seed, p->c0_update_seed, p->sk_t0);
            gpuErrchk(hipPeekAtLastError());
    }
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    return std::chrono::duration_cast<std::chrono::milliseconds>(end-begin).count()/ (double) iterations;
}



int main( int argc, const char* argv[] )
{
    parameters p;
    hipSetDevice(0);

    p.replicas = (unsigned int) atoll(argv[1]);
    p.skn_cols = (unsigned int) atoll(argv[2]);

    p.ts0= 2147483648/4;
    
    hipMalloc((void **) &p.sk_t0, p.replicas*p.skn_cols*sizeof(unsigned int));
    hipMemset(p.sk_t0, 0, p.replicas*p.skn_cols*sizeof(unsigned int));

    unsigned int* t0_c0 = readUArrayFromFile("./data.dump");
    hipMalloc((void **) &p.c0, p.ts0*sizeof(unsigned int));
    hipMemcpy(p.c0, t0_c0, sizeof(unsigned int)*p.ts0, hipMemcpyHostToDevice);


    boost::random::mt19937 gen(1337);
    unsigned int* c0_select_seed =  (unsigned int*) malloc(sizeof(unsigned int)*32);
    for(unsigned int i = 0; i < 32; i++){
       c0_select_seed[i] = gen();
    }

    unsigned int* c0_update_seed =  (unsigned int*) malloc(sizeof(unsigned int)*32);
    for(unsigned int i = 0; i < 32; i++){
       c0_update_seed[i] = gen();
    }

    p.c0_select_seed = (unsigned int*) cudaAllocAndCopy(c0_select_seed, 32*sizeof(unsigned int));
    p.c0_update_seed = (unsigned int*) cudaAllocAndCopy(c0_update_seed, 32*sizeof(unsigned int));

    double time = sketch_contruction(&p);
    std::cout << p.replicas << ";" << p.skn_cols << ";" << p.ts0*sizeof(unsigned int)*8.0 / (1000.0*1000.0*1000.0*time / 1000.0) << std::endl;

    int* res = (int*) malloc(p.replicas*p.skn_cols*sizeof(int));
    hipMemcpy(res, p.sk_t0, p.replicas*p.skn_cols*sizeof(int), hipMemcpyDeviceToHost);

    writeSArrayToFile("sketch.dump", res, p.replicas*p.skn_cols);
    
    //Debugging print
    for(int i = 0; i < p.replicas; i++){
        //std::cout << "Row: | " << i << std::endl;
        int row_sum = 0;
        for(int j = 0; j < p.skn_cols; j++){
            row_sum += res[i*p.skn_cols+ j];
            //std::cout << res[i*p.skn_cols+ j] << " | ";
        }
        //std::cout << "| Sum: " << row_sum << std::endl;
    }


    return 0;
}
