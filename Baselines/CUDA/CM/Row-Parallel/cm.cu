#include "hip/hip_runtime.h"
#pragma GCC diagnostic ignored "-Wignored-attributes"
#include <iostream>
#include <chrono>

#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_int_distribution.hpp>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ unsigned int parity(unsigned int x) {
    unsigned int y;
    y = x ^ (x >> 1);
    y = y ^ (y >> 2);
    y = y ^ (y >> 4);
    y = y ^ (y >> 8);
    y = y ^ (y >>16);
    return y & 1;
 }
 
 __device__ unsigned int nonlinear_h(unsigned int x) {
     return parity((x >> 0) | (x >> 1));
 }
 
 __device__ unsigned int is_set(unsigned int x, unsigned int pos) {
     return (x >> pos) & 1;
 }
 

 __device__ unsigned int h3(unsigned int x, unsigned int nsketches, unsigned int* seed) {
    unsigned int hash = 0;
    for(int i = 0; i < 32; i++){
        hash ^=  seed[i*nsketches]*is_set(x,i); 
    }
    return hash;
}


/* __device__ unsigned int h3(unsigned int x, unsigned int nsketches, unsigned long* seed) {
    unsigned long hash = 0;
    for(int i = 0; i < 16; i++){
        unsigned long seed_l = seed[i*nsketches];
        hash ^= seed_l & ((is_set(x,i*2)*0x00000000FFFFFFFFL) | (is_set(x, i*2+1)*0xFFFFFFFF00000000L)) ;
    }
    return ((unsigned int) hash ^ (unsigned int) (hash >> 32));
}*/

 __device__ int ech3(unsigned int v, unsigned int seed, unsigned int sbit){
     //First we compute the bitwise AND between the seed and the value
     //Aaaand here comes the parity
     int res = parity(v & seed) ^ nonlinear_h(v) ^ sbit ;
     return 2*res-1;
 }

 __global__ void construct_sketch(
    unsigned int skn_rows,
    unsigned int skn_cols,
    unsigned int n_values,
    unsigned int n_replicas,
    unsigned int* __restrict__ c0,
    unsigned int* __restrict__ c0_ls, 
    unsigned int* __restrict__ c0_ss,
    unsigned int* __restrict__ c0_select_seed,
    int* __restrict__ sketches) 
{
    unsigned int sseeds[32];

    //unsigned int global_size = gridDim.x * blockDim.x;
    unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;

    //unsigned full_work_set = skn_rows * n_replicas;
    unsigned int partition_size = (n_values-1)/n_replicas+1;

        unsigned int partition = global_id / skn_rows;
        unsigned int sketch = global_id % skn_rows;
        //unsigned int my_c0_ls = (sketch < skn_rows) ? c0_ls[sketch] : 0; 
        //unsigned int my_c0_ss = (sketch < skn_rows) ? is_set(c0_ss[sketch/32], sketch % 32) : 0;
        //unsigned int* my_c0_select_seed = (sketch < skn_rows) ? c0_select_seed + sketch : 0;
   
        if(sketch < skn_rows){
            for(int i = 0; i < 32; i++){
                sseeds[i] = c0_select_seed[sketch+i*skn_rows];
            }
        }


        if(sketch >= skn_rows) return;
        for(unsigned int i = partition*partition_size; i < (partition+1)*partition_size && i < n_values; i++){
                unsigned int select = 0;
                //for(int k = 0; k < 32; k++)  select ^= sseeds[k] * ((c0[i] >> k) & 1) ;
                for(int k = 0; k < 32; k++)  if(is_set(c0[i],k)) select ^= sseeds[k] ;

                //select = select >> (1 + __clz(skn_cols));
                select = select % skn_cols;

                //int update = ech3(c0[i],my_c0_ls,my_c0_ss);
                int update = 1;
                atomicAdd(&sketches[sketch*skn_cols+select], update);
        }
}

typedef struct{

    size_t skn_rows;
    size_t skn_cols;

    int* sk_t0;
    unsigned int ts0;

    unsigned int* c0;
    unsigned int* c0_lseed;
    unsigned int* c0_sseed;
    unsigned int* c0_select_seed;

} parameters;

void* cudaAllocAndCopy(void* hst_ptr, size_t size){
    void* d_ptr;
    hipMalloc((void **) &d_ptr, size);
    hipMemcpy(d_ptr, hst_ptr, size, hipMemcpyHostToDevice);
    return d_ptr;
}

void writeSArrayToFile(const char* filename, int* elements, size_t size){
    FILE *f1 = fopen(filename, "w");
    assert(f1 != NULL);
    
    fwrite(elements, sizeof(int), size, f1);
    fclose(f1);
}

unsigned int* readUArrayFromFile(const char* filename, size_t * filesize = NULL){
    FILE *f1 = fopen(filename, "rb");
    assert(f1 != NULL);
    fseek(f1, 0, SEEK_END);
    size_t fsize1 = ftell(f1);
    if(filesize) *filesize=fsize1;
    fseek(f1, 0, SEEK_SET);
    unsigned int* tab1 = (unsigned int*) malloc(fsize1);
    size_t x = fread(tab1, fsize1, 1, f1);
    fclose(f1);

    return tab1;
}

double sketch_contruction(parameters* p){
    size_t local = 32;
    int tot_SM = 0;
    hipDeviceGetAttribute(&tot_SM, hipDeviceAttributeMultiprocessorCount, 0);
    unsigned int target_utilization = tot_SM*2048;
    unsigned int n_partitions = target_utilization / p->skn_rows;
    unsigned int target_global_size = n_partitions * p->skn_rows;
    
    //We do not allow sketch sizes that exceed the target utilization
    assert(n_partitions > 0);

    //We do not allow sketch sizes that are not a multiple of the specified sketch size.
    //We can only cache memory accesses if all threads in a work group work to the same partition.
    //assert(p->skn_rows % local == 0);


    size_t global = target_global_size;
    //std::cout << "local: " << local << " global:" << global << " partitions: " << n_partitions << std::endl;

    auto begin = std::chrono::high_resolution_clock::now();
    int iterations = 1;

//    for(int i = 0; i < iterations; i++){
        construct_sketch<<<global/local, local>>>((unsigned int) p->skn_rows, (unsigned int) p->skn_cols, p->ts0, n_partitions, p->c0, p->c0_lseed, p->c0_sseed, p->c0_select_seed, p->sk_t0);
        gpuErrchk(hipPeekAtLastError());
        hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();
    return std::chrono::duration_cast<std::chrono::milliseconds>(end-begin).count()/(double) iterations;
}



int main( int argc, const char* argv[] )
{
    parameters p;
    hipSetDevice(0);

    p.skn_rows = (unsigned int) atoll(argv[1]);
    p.skn_cols = (unsigned int) atoll(argv[2]);

    p.ts0= 2147483648/4;
    
    hipMalloc((void **) &p.sk_t0, p.skn_rows*p.skn_cols*sizeof(int));
    hipMemset(p.sk_t0, 0, p.skn_rows*p.skn_cols*sizeof(int));

    unsigned int* t0_c0 = readUArrayFromFile("./data.dump");
    hipMalloc((void **) &p.c0, p.ts0*sizeof(unsigned int));
    hipMemcpy(p.c0, t0_c0, sizeof(unsigned int)*p.ts0, hipMemcpyHostToDevice);


    unsigned int* c0_lseed =  (unsigned int*) malloc(sizeof(unsigned int)*p.skn_rows);
    unsigned int* c0_sseed =  (unsigned int*) malloc(((p.skn_rows-1)/(sizeof(unsigned int)*8) +1)*sizeof(unsigned int));
    unsigned int* c0_select_seed =  (unsigned int*) malloc(sizeof(unsigned int)*32*p.skn_rows);

    boost::random::mt19937 gen(1338);
    for(unsigned int i = 0; i < p.skn_rows;  i++ ){
       c0_lseed[i] = gen();
    }
    for(unsigned int i = 0; i < 32*p.skn_rows;  i++ ){
       c0_select_seed[i] = gen();
    }
    for(unsigned int i = 0; i < ((p.skn_rows-1)/(sizeof(unsigned int)*8) +1);  i++ ){
       c0_sseed[i] = gen();
    }
    p.c0_lseed = (unsigned int*) cudaAllocAndCopy(c0_lseed, sizeof(unsigned int)*p.skn_rows);
    p.c0_select_seed = (unsigned int*) cudaAllocAndCopy(c0_select_seed, 32*p.skn_rows*sizeof(unsigned int));
    p.c0_sseed = (unsigned int*) cudaAllocAndCopy(c0_sseed, ((p.skn_rows-1)/(sizeof(unsigned int)*8) +1)*sizeof(unsigned int));

    double time = sketch_contruction(&p);
    //std::cout << "GPU Execution Time: " << time << std::endl;
    //std::cout << "Normalized execution time: " << ((float) time /p.skn_rows) << std::endl;
    //std::cout << "Throughput: "<< p.ts0*sizeof(unsigned int)*8.0 / (1000.0*1000.0*1000.0*time / 1000.0) << " gbps" << std::endl;
    //std::cout << "Normalized Throughput: "<< p.ts0*sizeof(unsigned int)*8.0*p.skn_rows / (1000.0*1000.0*1000.0*time/1000.0) << " gbps" << std::endl;
    std::cout << p.skn_rows << ";" << p.skn_cols << ";" << p.ts0*sizeof(unsigned int)*8.0 / (1000.0*1000.0*1000.0*time / 1000.0) << std::endl;

    int* res = (int*) malloc(p.skn_rows*p.skn_cols*sizeof(int));
    hipMemcpy(res, p.sk_t0, p.skn_rows*p.skn_cols*sizeof(int), hipMemcpyDeviceToHost);

    writeSArrayToFile("sketch.dump", res, p.skn_rows*p.skn_cols);
    /*for(int i = 0; i < p.skn_rows; i++){
        //std::cout << "Row: | " << i << std::endl;
        int row_sum = 0;
        for(int j = 0; j < p.skn_cols; j++){
            row_sum += res[i*p.skn_cols+ j];
            std::cout << res[i*p.skn_cols+ j] << " | ";
        }
        std::cout << "| Sum: " << row_sum << std::endl;
    }*/


    return 0;
}
