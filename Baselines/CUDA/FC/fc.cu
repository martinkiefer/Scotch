#include "hip/hip_runtime.h"
#pragma GCC diagnostic ignored "-Wignored-attributes"
#include <iostream>
#include <chrono>

#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_int_distribution.hpp>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ unsigned int parity(unsigned int x) {
    unsigned int y;
    y = x ^ (x >> 1);
    y = y ^ (y >> 2);
    y = y ^ (y >> 4);
    y = y ^ (y >> 8);
    y = y ^ (y >>16);
    return y & 1;
 }
 
 __device__ unsigned int nonlinear_h(unsigned int x) {
     return parity((x >> 0) | (x >> 1));
 }
 
 __device__ unsigned int is_set(unsigned int x, unsigned int pos) {
     return (x >> pos) & 1;
 }
 

 __device__ unsigned int h3(unsigned int x, unsigned int nsketches, unsigned int* seed) {
    unsigned int hash = 0;
    for(int i = 0; i < 32; i++){
        hash ^=  seed[i*nsketches]*is_set(x,i); 
    }
    return hash;
}

 __device__ unsigned long cw4(unsigned long a, unsigned long b1, unsigned long b2, unsigned long b3, unsigned long x) {
    unsigned int p = 2147483647;
    unsigned long res = a*x + b1;
    res = (res >> 31) + (res & p);
    for(int i = 0; i < 3; i++) res -= p * (res > p);


    res = res*x + b2;
    res = (res >> 31) + (res & p);
    for(int i = 0; i < 3; i++) res -= p * (res > p);

    res = res*x + b3;
    res = (res >> 31) + (res & p);
    for(int i = 0; i < 3; i++) res -= p * (res > p);

    return res;
}

 __global__ void construct_sketch(
    unsigned int replicas,
    unsigned int skn_cols,
    unsigned int n_values,
    unsigned int* __restrict__ c0,
    unsigned int* __restrict__ c0_select_seed,
    int* __restrict__ sketches) 
{
    unsigned int global_size = gridDim.x * blockDim.x;
    unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;



    unsigned long a = c0_select_seed[0];
    unsigned long b1 = c0_select_seed[1];
    unsigned long b2 = c0_select_seed[2]; 
    unsigned long b3 = c0_select_seed[3]; 
    unsigned int r = global_id % replicas;

    //Energy consumption endless loop. Comment in, if necessary.
    //while(1)
    for(unsigned int i = global_id; i < n_values; i += global_size){
            unsigned int select = cw4(a, b1, b2, b3, c0[i]);
            select = select % skn_cols;

            int update = 1;
            atomicAdd(&sketches[r*skn_cols+select], update);
    }
}

typedef struct{

    size_t replicas;
    size_t skn_cols;

    int* sk_t0;
    unsigned int ts0;

    unsigned int* c0;
    unsigned int* c0_select_seed;

} parameters;

void* cudaAllocAndCopy(void* hst_ptr, size_t size){
    void* d_ptr;
    hipMalloc((void **) &d_ptr, size);
    hipMemcpy(d_ptr, hst_ptr, size, hipMemcpyHostToDevice);
    return d_ptr;
}

void writeSArrayToFile(const char* filename, int* elements, size_t size){
    FILE *f1 = fopen(filename, "w");
    assert(f1 != NULL);
    
    fwrite(elements, sizeof(int), size, f1);
    fclose(f1);
}

unsigned int* readUArrayFromFile(const char* filename, size_t * filesize = NULL){
    FILE *f1 = fopen(filename, "rb");
    assert(f1 != NULL);
    fseek(f1, 0, SEEK_END);
    size_t fsize1 = ftell(f1);
    if(filesize) *filesize=fsize1;
    fseek(f1, 0, SEEK_SET);
    unsigned int* tab1 = (unsigned int*) malloc(fsize1);
    size_t x = fread(tab1, fsize1, 1, f1);
    fclose(f1);

    return tab1;
}

double sketch_contruction(parameters* p){
    size_t local = 64;
    int tot_SM = 0;
    int tot_tpsm = 0;
    hipDeviceGetAttribute(&tot_SM, hipDeviceAttributeMultiprocessorCount, 0);
    hipDeviceGetAttribute(&tot_tpsm, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
    unsigned int target_utilization = tot_SM*tot_tpsm;

    size_t global = target_utilization;

    auto begin = std::chrono::high_resolution_clock::now();
    int iterations = 1;
    for(int i = 0; i < iterations; i++){
            construct_sketch<<<global/local, local>>>((unsigned int) p->replicas, (unsigned int) p->skn_cols, p->ts0, p->c0, p->c0_select_seed, p->sk_t0);
            gpuErrchk(hipPeekAtLastError());
    }
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    return std::chrono::duration_cast<std::chrono::milliseconds>(end-begin).count()/ (double) iterations;
}



int main( int argc, const char* argv[] )
{
    parameters p;
    hipSetDevice(0);

    p.replicas = (unsigned int) atoll(argv[1]);
    p.skn_cols = (unsigned int) atoll(argv[2]);

    p.ts0= 2147483648/4;
    
    hipMalloc((void **) &p.sk_t0, p.replicas*p.skn_cols*sizeof(int));
    hipMemset(p.sk_t0, 0, p.replicas*p.skn_cols*sizeof(int));

    unsigned int* t0_c0 = readUArrayFromFile("./data.dump");
    hipMalloc((void **) &p.c0, p.ts0*sizeof(unsigned int));
    hipMemcpy(p.c0, t0_c0, sizeof(unsigned int)*p.ts0, hipMemcpyHostToDevice);


    unsigned int* c0_select_seed =  (unsigned int*) malloc(sizeof(unsigned int)*4);

    boost::random::mt19937 gen(1338);
    for(unsigned int i = 0; i < 4; i++){
       c0_select_seed[i] = gen() % 2147483647;
    }

    p.c0_select_seed = (unsigned int*) cudaAllocAndCopy(c0_select_seed, 4*p.replicas*sizeof(unsigned int));

    double time = sketch_contruction(&p);
    std::cout << p.replicas << ";" << p.skn_cols << ";" << p.ts0*sizeof(unsigned int)*8.0 / (1000.0*1000.0*1000.0*time / 1000.0) << std::endl;

    int* res = (int*) malloc(p.replicas*p.skn_cols*sizeof(int));
    hipMemcpy(res, p.sk_t0, p.replicas*p.skn_cols*sizeof(int), hipMemcpyDeviceToHost);

    writeSArrayToFile("sketch.dump", res, p.replicas*p.skn_cols);
    
    //Debugging print
/*    for(int i = 0; i < p.replicas; i++){
        //std::cout << "Row: | " << i << std::endl;
        int row_sum = 0;
        for(int j = 0; j < p.skn_cols; j++){
            row_sum += res[i*p.skn_cols+ j];
            std::cout << res[i*p.skn_cols+ j] << " | ";
        }
        std::cout << "| Sum: " << row_sum << std::endl;
    }
*/

    return 0;
}
