#include "hip/hip_runtime.h"
#pragma GCC diagnostic ignored "-Wignored-attributes"
#include <iostream>
#include <chrono>

#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_int_distribution.hpp>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ unsigned int parity(unsigned int x) {
    unsigned int y;
    y = x ^ (x >> 1);
    y = y ^ (y >> 2);
    y = y ^ (y >> 4);
    y = y ^ (y >> 8);
    y = y ^ (y >>16);
    return y & 1;
 }
 
 __device__ unsigned int nonlinear_h(unsigned int x) {
     return parity(((x << 0) | (x << 1)) & 0xAAAAAAAA );
 }
 
 __device__ unsigned int is_set(unsigned int x, unsigned int pos) {
     return (x >> pos) & 1;
 }
 

 __device__ unsigned int h3(unsigned int x, unsigned int nsketches, unsigned int* seed) {
    unsigned int hash = 0;
    for(int i = 0; i < 32; i++){
        hash ^=  seed[i*nsketches]*is_set(x,i); 
    }
    return hash;
}


 __device__ int ech3(unsigned int v, unsigned int seed, unsigned int sbit){
     //First we compute the bitwise AND between the seed and the value
     //Aaaand here comes the parity
     int res = parity(v & seed) ^ nonlinear_h(v) ^ sbit ;
     return 2*res-1;
 }
 __global__ void construct_sketch_coalesced(
    unsigned int skn_rows,
    unsigned int skn_cols,
    unsigned int n_values,
    unsigned int n_replicas,
    unsigned int* __restrict__ c0,
    unsigned int* __restrict__ c0_ls, 
    unsigned int* __restrict__ c0_ss,
    unsigned int* __restrict__ c0_select_seed,
    int* __restrict__ sketches) 
{
    unsigned int sseeds[32];
    unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int partition = global_id % n_replicas;
    unsigned int sketch = global_id / n_replicas;
    unsigned int my_c0_ls = c0_ls[sketch]; 
    unsigned int my_c0_ss = is_set(c0_ss[sketch/32], sketch % 32);

    for(int i = 0; i < 32; i++){
        sseeds[i] = c0_select_seed[sketch+i*skn_rows];
    }

    for(unsigned int i = partition; i < n_values; i += n_replicas){
        unsigned int select = 0;
        for(int k = 0; k < 32; k++)  if(is_set(c0[i],k)) select ^= sseeds[k] ;

        select = select % skn_cols;
        int update = ech3(c0[i],my_c0_ls,my_c0_ss);
        atomicAdd(&sketches[sketch*skn_cols+select], update);
    }
}

 __global__ void construct_sketch(
    unsigned int skn_rows,
    unsigned int skn_cols,
    unsigned int n_values,
    unsigned int n_replicas,
    unsigned int* __restrict__ c0,
    unsigned int* __restrict__ c0_ls, 
    unsigned int* __restrict__ c0_ss,
    unsigned int* __restrict__ c0_select_seed,
    int* __restrict__ sketches) 
{
    unsigned int sseeds[32];
    unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int partition_size = (n_values-1)/n_replicas+1;

    unsigned int partition = global_id / skn_rows;
    unsigned int sketch = global_id % skn_rows;
    unsigned int my_c0_ls = c0_ls[sketch]; 
    unsigned int my_c0_ss = is_set(c0_ss[sketch/32], sketch % 32);

    for(int i = 0; i < 32; i++){
        sseeds[i] = c0_select_seed[sketch+i*skn_rows];
    }

    //Energy consumption endless loop. Comment in, if necessary.
    //while(1)
    for(unsigned int i = partition*partition_size; i < (partition+1)*partition_size && i < n_values; i++){
        unsigned int select = 0;
        for(int k = 0; k < 32; k++)  if(is_set(c0[i],k)) select ^= sseeds[k] ;

        select = select % skn_cols;
        int update = ech3(c0[i],my_c0_ls,my_c0_ss);
        atomicAdd(&sketches[sketch*skn_cols+select], update);
    }
}

typedef struct{

    size_t skn_rows;
    size_t skn_cols;

    int* sk_t0;
    unsigned int ts0;

    unsigned int* c0;
    unsigned int* c0_lseed;
    unsigned int* c0_sseed;
    unsigned int* c0_select_seed;

} parameters;

void* cudaAllocAndCopy(void* hst_ptr, size_t size){
    void* d_ptr;
    hipMalloc((void **) &d_ptr, size);
    hipMemcpy(d_ptr, hst_ptr, size, hipMemcpyHostToDevice);
    return d_ptr;
}

void writeSArrayToFile(const char* filename, int* elements, size_t size){
    FILE *f1 = fopen(filename, "w");
    assert(f1 != NULL);
    
    fwrite(elements, sizeof(int), size, f1);
    fclose(f1);
}

unsigned int* readUArrayFromFile(const char* filename, size_t * filesize = NULL){
    FILE *f1 = fopen(filename, "rb");
    assert(f1 != NULL);
    fseek(f1, 0, SEEK_END);
    size_t fsize1 = ftell(f1);
    if(filesize) *filesize=fsize1;
    fseek(f1, 0, SEEK_SET);
    unsigned int* tab1 = (unsigned int*) malloc(fsize1);
    size_t x = fread(tab1, fsize1, 1, f1);
    fclose(f1);

    return tab1;
}

double sketch_contruction(parameters* p){
    size_t local = 64;
    int tot_SM = 0;
    int tot_tpsm = 0;
    hipDeviceGetAttribute(&tot_SM, hipDeviceAttributeMultiprocessorCount, 0);
    hipDeviceGetAttribute(&tot_tpsm, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
    unsigned int target_utilization = tot_SM*tot_tpsm;
    unsigned int n_partitions = target_utilization / p->skn_rows;
    unsigned int target_global_size = n_partitions * p->skn_rows;
    
    //We do not allow sketch sizes that exceed the target utilization
    assert(n_partitions > 0);

    size_t global = target_global_size;

    auto begin = std::chrono::high_resolution_clock::now();
    construct_sketch<<<global/local, local>>>((unsigned int) p->skn_rows, (unsigned int) p->skn_cols, p->ts0, n_partitions, p->c0, p->c0_lseed, p->c0_sseed, p->c0_select_seed, p->sk_t0);
    gpuErrchk(hipPeekAtLastError());
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    return std::chrono::duration_cast<std::chrono::milliseconds>(end-begin).count();
}



int main( int argc, const char* argv[] )
{
    parameters p;
    hipSetDevice(0);

    p.skn_rows = (unsigned int) atoll(argv[1]);
    p.skn_cols = (unsigned int) atoll(argv[2]);

    p.ts0= 2147483648/4;
    
    hipMalloc((void **) &p.sk_t0, p.skn_rows*p.skn_cols*sizeof(int));
    hipMemset(p.sk_t0, 0, p.skn_rows*p.skn_cols*sizeof(int));

    unsigned int* t0_c0 = readUArrayFromFile("./data.dump");
    hipMalloc((void **) &p.c0, p.ts0*sizeof(unsigned int));
    hipMemcpy(p.c0, t0_c0, sizeof(unsigned int)*p.ts0, hipMemcpyHostToDevice);


    unsigned int* c0_lseed =  (unsigned int*) malloc(sizeof(unsigned int)*p.skn_rows);
    unsigned int* c0_sseed =  (unsigned int*) malloc(((p.skn_rows-1)/(sizeof(unsigned int)*8) +1)*sizeof(unsigned int));
    unsigned int* c0_select_seed =  (unsigned int*) malloc(sizeof(unsigned int)*32*p.skn_rows);

    boost::random::mt19937 gen(1338);
    for(unsigned int i = 0; i < p.skn_rows;  i++ ){
       c0_lseed[i] = gen();
    }
    for(unsigned int i = 0; i < 32*p.skn_rows;  i++ ){
       c0_select_seed[i] = gen();
    }
    for(unsigned int i = 0; i < ((p.skn_rows-1)/(sizeof(unsigned int)*8) +1);  i++ ){
       c0_sseed[i] = gen();
    }
    p.c0_lseed = (unsigned int*) cudaAllocAndCopy(c0_lseed, sizeof(unsigned int)*p.skn_rows);
    p.c0_select_seed = (unsigned int*) cudaAllocAndCopy(c0_select_seed, 32*p.skn_rows*sizeof(unsigned int));
    p.c0_sseed = (unsigned int*) cudaAllocAndCopy(c0_sseed, ((p.skn_rows-1)/(sizeof(unsigned int)*8) +1)*sizeof(unsigned int));

    double time = sketch_contruction(&p);
    std::cout << p.skn_rows << ";" << p.skn_cols << ";" << p.ts0*sizeof(unsigned int)*8.0 / (1000.0*1000.0*1000.0*time / 1000.0) << std::endl;

    int* res = (int*) malloc(p.skn_rows*p.skn_cols*sizeof(int));
    hipMemcpy(res, p.sk_t0, p.skn_rows*p.skn_cols*sizeof(int), hipMemcpyDeviceToHost);

    writeSArrayToFile("sketch.dump", res, p.skn_rows*p.skn_cols);

    return 0;
}
