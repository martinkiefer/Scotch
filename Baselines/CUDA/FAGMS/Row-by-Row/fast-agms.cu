#include "hip/hip_runtime.h"
#pragma GCC diagnostic ignored "-Wignored-attributes"
#include <iostream>
#include <chrono>

#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_int_distribution.hpp>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ unsigned int parity(unsigned int x) {
    unsigned int y;
    y = x ^ (x >> 1);
    y = y ^ (y >> 2);
    y = y ^ (y >> 4);
    y = y ^ (y >> 8);
    y = y ^ (y >>16);
    return y & 1;
 }
 
 __device__ unsigned int nonlinear_h(unsigned int x) {
     return parity((x >> 0) | (x >> 1));
 }
 
 __device__ unsigned int is_set(unsigned int x, unsigned int pos) {
     return (x >> pos) & 1;
 }
 

 __device__ unsigned int h3(unsigned int x, unsigned int nsketches, unsigned int* seed) {
    unsigned int hash = 0;
    for(int i = 0; i < 32; i++){
        hash ^=  seed[i*nsketches]*is_set(x,i); 
    }
    return hash;
}


/* __device__ unsigned int h3(unsigned int x, unsigned int nsketches, unsigned long* seed) {
    unsigned long hash = 0;
    for(int i = 0; i < 16; i++){
        unsigned long seed_l = seed[i*nsketches];
        hash ^= seed_l & ((is_set(x,i*2)*0x00000000FFFFFFFFL) | (is_set(x, i*2+1)*0xFFFFFFFF00000000L)) ;
    }
    return ((unsigned int) hash ^ (unsigned int) (hash >> 32));
}*/

 __device__ int ech3(unsigned int v, unsigned int seed, unsigned int sbit){
     //First we compute the bitwise AND between the seed and the value
     //Aaaand here comes the parity
     int res = parity(v & seed) ^ nonlinear_h(v) ^ sbit ;
     return 2*res-1;
 }

 __global__ void construct_sketch(
    unsigned int r,
    unsigned int skn_cols,
    unsigned int n_values,
    unsigned int* __restrict__ c0,
    unsigned int* __restrict__ c0_ls, 
    unsigned int* __restrict__ c0_ss,
    unsigned int* __restrict__ c0_select_seed,
    int* __restrict__ sketches) 
{
    unsigned int sseeds[32];

    unsigned int global_size = gridDim.x * blockDim.x;
    unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;


    unsigned int my_c0_ls = c0_ls[r]; 
    unsigned int my_c0_ss = is_set(c0_ss[r/32], r % 32);
    //unsigned int* my_c0_select_seed = (sketch < skn_rows) ? c0_select_seed + sketch : 0;


    for(int i = 0; i < 32; i++){
        sseeds[i] = c0_select_seed[r*32+i];
    }

    for(unsigned int i = global_id; i < n_values; i += global_size){
            unsigned int select = 0;
            for(int k = 0; k < 32; k++)  if(is_set(c0[i],k)) select ^= sseeds[k] ;
            select = select % skn_cols;

            int update = ech3(c0[i],my_c0_ls,my_c0_ss);
            //int update = 1;
            atomicAdd(&sketches[r*skn_cols+select], update);
    }
}

typedef struct{

    size_t skn_rows;
    size_t skn_cols;

    int* sk_t0;
    unsigned int ts0;

    unsigned int* c0;
    unsigned int* c0_lseed;
    unsigned int* c0_sseed;
    unsigned int* c0_select_seed;

} parameters;

void* cudaAllocAndCopy(void* hst_ptr, size_t size){
    void* d_ptr;
    hipMalloc((void **) &d_ptr, size);
    hipMemcpy(d_ptr, hst_ptr, size, hipMemcpyHostToDevice);
    return d_ptr;
}

void writeSArrayToFile(const char* filename, int* elements, size_t size){
    FILE *f1 = fopen(filename, "w");
    assert(f1 != NULL);
    
    fwrite(elements, sizeof(int), size, f1);
    fclose(f1);
}

unsigned int* readUArrayFromFile(const char* filename, size_t * filesize = NULL){
    FILE *f1 = fopen(filename, "rb");
    assert(f1 != NULL);
    fseek(f1, 0, SEEK_END);
    size_t fsize1 = ftell(f1);
    if(filesize) *filesize=fsize1;
    fseek(f1, 0, SEEK_SET);
    unsigned int* tab1 = (unsigned int*) malloc(fsize1);
    size_t x = fread(tab1, fsize1, 1, f1);
    fclose(f1);

    return tab1;
}

double sketch_contruction(parameters* p){
    size_t local = 32;
    int tot_SM = 0;
    hipDeviceGetAttribute(&tot_SM, hipDeviceAttributeMultiprocessorCount, 0);
    unsigned int target_utilization = tot_SM*2048;
    size_t global = target_utilization;

    auto begin = std::chrono::high_resolution_clock::now();
    int iterations = 1;
    for(int i = 0; i < iterations; i++){
        for(unsigned int r = 0; r < p->skn_rows; r++){
            construct_sketch<<<global/local, local>>>(r, (unsigned int) p->skn_cols, p->ts0, p->c0, p->c0_lseed, p->c0_sseed, p->c0_select_seed, p->sk_t0);
            gpuErrchk(hipPeekAtLastError());
        }
    }
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    return std::chrono::duration_cast<std::chrono::milliseconds>(end-begin).count()/ (double) iterations;
}



int main( int argc, const char* argv[] )
{
    parameters p;
    hipSetDevice(0);

    p.skn_rows = (unsigned int) atoll(argv[1]);
    p.skn_cols = (unsigned int) atoll(argv[2]);

    p.ts0= 2147483648/4;
    
    hipMalloc((void **) &p.sk_t0, p.skn_rows*p.skn_cols*sizeof(int));
    hipMemset(p.sk_t0, 0, p.skn_rows*p.skn_cols*sizeof(int));

    unsigned int* t0_c0 = readUArrayFromFile("./data.dump");
    hipMalloc((void **) &p.c0, p.ts0*sizeof(unsigned int));
    hipMemcpy(p.c0, t0_c0, sizeof(unsigned int)*p.ts0, hipMemcpyHostToDevice);


    unsigned int* c0_lseed =  (unsigned int*) malloc(sizeof(unsigned int)*p.skn_rows);
    unsigned int* c0_sseed =  (unsigned int*) malloc(((p.skn_rows-1)/(sizeof(unsigned int)*8) +1)*sizeof(unsigned int));
    unsigned int* c0_select_seed =  (unsigned int*) malloc(sizeof(unsigned int)*32*32*p.skn_rows);

    boost::random::mt19937 gen(1338);
    for(unsigned int i = 0; i < p.skn_rows;  i++ ){
       c0_lseed[i] = gen();
    }
    for(unsigned int i = 0; i < p.skn_rows*32*32; i++){
       c0_select_seed[i] = gen();
    }
    for(unsigned int i = 0; i < ((p.skn_rows-1)/(sizeof(unsigned int)*8) +1);  i++ ){
       c0_sseed[i] = gen();
    }
    p.c0_lseed = (unsigned int*) cudaAllocAndCopy(c0_lseed, sizeof(unsigned int)*p.skn_rows);
    p.c0_select_seed = (unsigned int*) cudaAllocAndCopy(c0_select_seed, p.skn_rows*32*32*sizeof(unsigned int));
    p.c0_sseed = (unsigned int*) cudaAllocAndCopy(c0_sseed, ((p.skn_rows-1)/(sizeof(unsigned int)*8) +1)*sizeof(unsigned int));

    double time = sketch_contruction(&p);
    std::cout << p.skn_rows << ";" << p.skn_cols << ";" << p.ts0*sizeof(unsigned int)*8.0 / (1000.0*1000.0*1000.0*time / 1000.0) << std::endl;

    int* res = (int*) malloc(p.skn_rows*p.skn_cols*sizeof(int));
    hipMemcpy(res, p.sk_t0, p.skn_rows*p.skn_cols*sizeof(int), hipMemcpyDeviceToHost);

    writeSArrayToFile("sketch.dump", res, p.skn_rows*p.skn_cols);
    
    /*Debugging print
    for(int i = 0; i < p.skn_rows; i++){
        //std::cout << "Row: | " << i << std::endl;
        int row_sum = 0;
        for(int j = 0; j < p.skn_cols; j++){
            row_sum += res[i*p.skn_cols+ j];
            std::cout << res[i*p.skn_cols+ j] << " | ";
        }
        std::cout << "| Sum: " << row_sum << std::endl;
    }*/


    return 0;
}
